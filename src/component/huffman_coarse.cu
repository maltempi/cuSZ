/**
 * @file huffman_coarse.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2021-12-17
 * (created) 2020-04-24 (rev1) 2021-09-05 (rev2) 2021-12-29
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * @copyright (C) 2021 by Washington State University, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include "huffman_coarse.cuh"

#define HUFFCOARSE(E, ETF, H, M) \
    template class cusz::HuffmanCoarse<ErrCtrlTrait<E, ETF>::type, HuffTrait<H>::type, MetadataTrait<M>::type>;

HUFFCOARSE(2, false, 4, 4)  // deprecated
HUFFCOARSE(2, false, 8, 4)  // deprecated
HUFFCOARSE(4, false, 4, 4)  // deprecated
HUFFCOARSE(4, false, 8, 4)  // deprecated

HUFFCOARSE(4, true, 4, 4)  // float
HUFFCOARSE(4, true, 8, 4)  // float

#undef HUFFCOARSE
