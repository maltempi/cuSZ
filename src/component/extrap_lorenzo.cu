/**
 * @file extrap_lorenzo.cu
 * @author Jiannan Tian
 * @brief A high-level LorenzoND wrapper. Allocations are explicitly out of called functions.
 * @version 0.3
 * @date 2021-06-16
 * (rev.1) 2021-09-18 (rev.2) 2022-01-10
 *
 * (C) 2021 by Washington State University, Argonne National Laboratory
 *
 */

#include "extrap_lorenzo.cuh"

template class cusz::PredictorLorenzo<float, uint16_t, float>;
template class cusz::PredictorLorenzo<float, uint32_t, float>;
template class cusz::PredictorLorenzo<float, float, float>;
