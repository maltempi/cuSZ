/**
 * @file kernel_cuda.cc
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2022-07-24
 *
 * (C) 2022 by Washington State University, Argonne National Laboratory
 *
 */

#include "kernel/kernel_cuda.h"
#include "kernel/launch_prediction.cuh"

#define CONSTRUCT_LORENZOI(Tliteral, Eliteral, FPliteral, T, E, FP)                                            \
    cusz_error_status launch_construct_LorenzoI_T##Tliteral##_E##Eliteral##_FP##FPliteral(                     \
        bool NO_R_SEPARATE, T* const data, dim3 const len3, T* const anchor, dim3 const placeholder_1,         \
        E* const errctrl, dim3 const placeholder_2, double const eb, int const radius, float* time_elapsed,    \
        hipStream_t stream)                                                                                   \
    {                                                                                                          \
        if (NO_R_SEPARATE)                                                                                     \
            launch_construct_LorenzoI<T, E, FP, true>(                                                         \
                data, len3, anchor, placeholder_1, errctrl, placeholder_2, eb, radius, *time_elapsed, stream); \
        else                                                                                                   \
            launch_construct_LorenzoI<T, E, FP, false>(                                                        \
                data, len3, anchor, placeholder_1, errctrl, placeholder_2, eb, radius, *time_elapsed, stream); \
        return CUSZ_SUCCESS;                                                                                   \
    }

CONSTRUCT_LORENZOI(fp32, ui8, fp32, float, uint8_t, float);
CONSTRUCT_LORENZOI(fp32, ui16, fp32, float, uint16_t, float);
CONSTRUCT_LORENZOI(fp32, ui32, fp32, float, uint32_t, float);
CONSTRUCT_LORENZOI(fp32, fp32, fp32, float, float, float);

#undef CONSTRUCT_LORENZOI

#define RECONSTRUCT_LORENZOI(Tliteral, Eliteral, FPliteral, T, E, FP)                                         \
    cusz_error_status launch_reconstruct_LorenzoI_T##Tliteral##_E##Eliteral##_FP##FPliteral(                  \
        T* xdata, dim3 const len3, T* anchor, dim3 const placeholder_1, E* errctrl, dim3 const placeholder_2, \
        double const eb, int const radius, float* time_elapsed, hipStream_t stream)                          \
    {                                                                                                         \
        launch_reconstruct_LorenzoI<T, E, FP>(                                                                \
            xdata, len3, anchor, placeholder_1, errctrl, placeholder_2, eb, radius, *time_elapsed, stream);   \
        return CUSZ_SUCCESS;                                                                                  \
    }

RECONSTRUCT_LORENZOI(fp32, ui8, fp32, float, uint8_t, float);
RECONSTRUCT_LORENZOI(fp32, ui16, fp32, float, uint16_t, float);
RECONSTRUCT_LORENZOI(fp32, ui32, fp32, float, uint32_t, float);
RECONSTRUCT_LORENZOI(fp32, fp32, fp32, float, float, float);

#undef RECONSTRUCT_LORENZOI

#define CONSTRUCT_SPLINE3(Tliteral, Eliteral, FPliteral, T, E, FP)                                                   \
    cusz_error_status launch_construct_Spline3_T##Tliteral##_E##Eliteral##_FP##FPliteral(                            \
        bool NO_R_SEPARATE, T* data, dim3 const len3, T* anchor, dim3 const an_len3, E* errctrl, dim3 const ec_len3, \
        double const eb, int const radius, float* time_elapsed, hipStream_t stream)                                 \
    {                                                                                                                \
        if (NO_R_SEPARATE)                                                                                           \
            launch_construct_Spline3<T, E, FP, true>(                                                                \
                data, len3, anchor, an_len3, errctrl, ec_len3, eb, radius, *time_elapsed, stream);                   \
        else                                                                                                         \
            launch_construct_Spline3<T, E, FP, false>(                                                               \
                data, len3, anchor, an_len3, errctrl, ec_len3, eb, radius, *time_elapsed, stream);                   \
        return CUSZ_SUCCESS;                                                                                         \
    }

CONSTRUCT_SPLINE3(fp32, ui8, fp32, float, uint8_t, float);
CONSTRUCT_SPLINE3(fp32, ui16, fp32, float, uint16_t, float);
CONSTRUCT_SPLINE3(fp32, ui32, fp32, float, uint32_t, float);
CONSTRUCT_SPLINE3(fp32, fp32, fp32, float, float, float);

#undef CONSTRUCT_SPLINE3

#define RECONSTRUCT_SPLINE3(Tliteral, Eliteral, FPliteral, T, E, FP)                                               \
    cusz_error_status launch_reconstruct_Spline3_T##Tliteral##_E##Eliteral##_FP##FPliteral(                        \
        T* xdata, dim3 const len3, T* anchor, dim3 const an_len3, E* errctrl, dim3 const ec_len3, double const eb, \
        int const radius, float* time_elapsed, hipStream_t stream)                                                \
    {                                                                                                              \
        launch_reconstruct_Spline3<T, E, FP>(                                                                      \
            xdata, len3, anchor, an_len3, errctrl, ec_len3, eb, radius, *time_elapsed, stream);                    \
        return CUSZ_SUCCESS;                                                                                       \
    }

RECONSTRUCT_SPLINE3(fp32, ui8, fp32, float, uint8_t, float);
RECONSTRUCT_SPLINE3(fp32, ui16, fp32, float, uint16_t, float);
RECONSTRUCT_SPLINE3(fp32, ui32, fp32, float, uint32_t, float);
RECONSTRUCT_SPLINE3(fp32, fp32, fp32, float, float, float);

#undef RECONSTRUCT_SPLINE3
